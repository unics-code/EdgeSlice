#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "dropout_layer.h"
#include "hip/hip_runtime.h"
#include "utils.h"
}

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale, int ix, int iy, int iz)
{
	int bx = blockIdx.x + ix;
	int by = blockIdx.y + iy;
	int bz = blockIdx.z + iz;

    int id = (bx + by*gridDim.x) * blockDim.x + threadIdx.x;
    if(id < size) input[id] = (rand[id] < prob) ? 0 : input[id]*scale;
}

void forward_dropout_layer_gpu(dropout_layer layer, network net)
{
    if (!net.train) return;
    int size = layer.inputs*layer.batch;
    cuda_random(layer.rand_gpu, size);
    /*
    int i;
    for(i = 0; i < size; ++i){
        layer.rand[i] = rand_uniform();
    }
    cuda_push_array(layer.rand_gpu, layer.rand, size);
    */

	if (VIRTUALGPU)
	{
		dim3 total_blocks = cuda_gridsize(size); int quato=QUATO;

		for (int iz=0;iz<total_blocks.z;iz++)
		{
			for (int iy=0;iy<total_blocks.y;iy++)
			{
				int ix = 0;

				for (ix=0;ix<int(total_blocks.x/quato);ix++)
					yoloswag420blazeit360noscope<<<quato, BLOCK>>>(net.input_gpu, size, layer.rand_gpu, layer.probability, layer.scale, ix*quato, iy, iz);

				// if the iteration is not integer, run last time with fixed number blocks
				if (double(total_blocks.x) - quato * int(double(total_blocks.x)/quato) > 0)
					yoloswag420blazeit360noscope<<<total_blocks.x - quato * ix, BLOCK>>>(net.input_gpu, size, layer.rand_gpu, layer.probability, layer.scale, ix*quato, iy, iz);
			}
		}
	}
	else
	{
		yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(net.input_gpu, size, layer.rand_gpu, layer.probability, layer.scale, 0, 0, 0);
	}

    // yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(net.input_gpu, size, layer.rand_gpu, layer.probability, layer.scale, 0, 0, 0);
    check_error(hipPeekAtLastError());
}

void backward_dropout_layer_gpu(dropout_layer layer, network net)
{
    if(!net.delta_gpu) return;
    int size = layer.inputs*layer.batch;

	if (VIRTUALGPU)
	{
		dim3 total_blocks = cuda_gridsize(size); int quato=QUATO;

		for (int iz=0;iz<total_blocks.z;iz++)
		{
			for (int iy=0;iy<total_blocks.y;iy++)
			{
				int ix = 0;

				for (ix=0;ix<int(total_blocks.x/quato);ix++)
					yoloswag420blazeit360noscope<<<quato, BLOCK>>>(net.delta_gpu, size, layer.rand_gpu, layer.probability, layer.scale, ix*quato, iy, iz);

				// if the iteration is not integer, run last time with fixed number blocks
				if (double(total_blocks.x) - quato * int(double(total_blocks.x)/quato) > 0)
					yoloswag420blazeit360noscope<<<total_blocks.x - quato * ix, BLOCK>>>(net.delta_gpu, size, layer.rand_gpu, layer.probability, layer.scale, ix*quato, iy, iz);
			}
		}
	}
	else
	{
		yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(net.delta_gpu, size, layer.rand_gpu, layer.probability, layer.scale, 0, 0, 0);
	}

    // yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(net.delta_gpu, size, layer.rand_gpu, layer.probability, layer.scale, 0, 0, 0);
    check_error(hipPeekAtLastError());
}
