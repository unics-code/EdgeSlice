#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "avgpool_layer.h"
#include "hip/hip_runtime.h"
}

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c,
		float *input, float *output, int ix, int iy, int iz) {
	int bx = blockIdx.x + ix;
	int by = blockIdx.y + iy;
	int bz = blockIdx.z + iz;

	int id = (bx + by * gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= n)
		return;

	int k = id % c;
	id /= c;
	int b = id;

	int i;
	int out_index = (k + c * b);
	output[out_index] = 0;
	for (i = 0; i < w * h; ++i) {
		int in_index = i + h * w * (k + b * c);
		output[out_index] += input[in_index];
	}
	output[out_index] /= w * h;
}

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c,
		float *in_delta, float *out_delta, int ix, int iy, int iz) {
	int bx = blockIdx.x + ix;
	int by = blockIdx.y + iy;
	int bz = blockIdx.z + iz;

	int id = (bx + by * gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= n)
		return;

	int k = id % c;
	id /= c;
	int b = id;

	int i;
	int out_index = (k + c * b);
	for (i = 0; i < w * h; ++i) {
		int in_index = i + h * w * (k + b * c);
		in_delta[in_index] += out_delta[out_index] / (w * h);
	}
}

extern "C" void forward_avgpool_layer_gpu(avgpool_layer layer, network net) {
	size_t n = layer.c * layer.batch;

	if (VIRTUALGPU) {
		dim3 total_blocks = cuda_gridsize(n); int quato=QUATO;

		for (int iz = 0; iz < total_blocks.z; iz++) {
			for (int iy = 0; iy < total_blocks.y; iy++) {
				int ix = 0;

				for (ix = 0; ix < int(total_blocks.x / quato); ix++)
					forward_avgpool_layer_kernel<<<quato, BLOCK>>>(n, layer.w,
							layer.h, layer.c, net.input_gpu, layer.output_gpu,
							ix * quato, iy, iz);

				// if the iteration is not integer, run last time with fixed number blocks
				if (double(total_blocks.x)
						- quato * int(double(total_blocks.x) / quato) > 0)
					forward_avgpool_layer_kernel<<<total_blocks.x - quato * ix,
							BLOCK>>>(n, layer.w, layer.h, layer.c,
							net.input_gpu, layer.output_gpu, ix * quato, iy,
							iz);
			}
		}
	} else {
		forward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK>>>(n, layer.w,
				layer.h, layer.c, net.input_gpu, layer.output_gpu, 0, 0, 0);
	}

	//forward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK>>>(n, layer.w, layer.h, layer.c, net.input_gpu, layer.output_gpu);
	check_error(hipPeekAtLastError());
}

extern "C" void backward_avgpool_layer_gpu(avgpool_layer layer, network net) {
	size_t n = layer.c * layer.batch;

	if (VIRTUALGPU) {
		dim3 total_blocks = cuda_gridsize(n); int quato=QUATO;

		for (int iz = 0; iz < total_blocks.z; iz++) {
			for (int iy = 0; iy < total_blocks.y; iy++) {
				int ix = 0;

				for (ix = 0; ix < int(total_blocks.x / quato); ix++)
					backward_avgpool_layer_kernel<<<quato, BLOCK>>>(n, layer.w,
							layer.h, layer.c, net.delta_gpu, layer.delta_gpu,
							ix * quato, iy, iz);

				// if the iteration is not integer, run last time with fixed number blocks
				if (double(total_blocks.x)
						- quato * int(double(total_blocks.x) / quato) > 0)
					backward_avgpool_layer_kernel<<<total_blocks.x - quato * ix,
							BLOCK>>>(n, layer.w, layer.h, layer.c,
							net.delta_gpu, layer.delta_gpu, ix * quato, iy, iz);
			}
		}
	} else {
		backward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK>>>(n, layer.w,
				layer.h, layer.c, net.delta_gpu, layer.delta_gpu, 0, 0, 0);
	}

	// backward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK>>>(n, layer.w, layer.h, layer.c, net.delta_gpu, layer.delta_gpu);
	check_error(hipPeekAtLastError());
}

